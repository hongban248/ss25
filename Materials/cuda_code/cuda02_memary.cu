#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void add_array(float *a)
{
	int i=threadIdx.x;
	a[i]=a[i]+1;
}

int main()
{
	int N=24;
	int i;

	float *dx=NULL;
	float *hx=NULL;

	int nbyte=N*sizeof(float);

	hipMalloc((void **)&dx,nbyte);
	
	if (dx==NULL){
		printf("cuda malloc fail!!");
		return -1;
	}
	printf("cuda melloc success!\n");
	
//	hx=(float *)malloc(nbyte);
	hipHostMalloc((void **)&hx,nbyte, hipHostMallocDefault);
	if (hx==NULL){
		printf("ram melloc fail!\n");
		return -2;
	}
	printf("ram melloc success!!\n");
	
	for(i=0;i<N;i++)
	{
		hx[i]=i;
		printf("%lf ",hx[i]);
	}
	printf("\n");
	
	hipMemcpy(dx,hx,nbyte,hipMemcpyHostToDevice);
	add_array<<<i,N>>>(dx);
	hipDeviceSynchronize();

	hipMemcpy(hx,dx,nbyte,hipMemcpyDeviceToHost);
	
	printf("N===%d\n",N);
	for (i=0;i<N;i++)
	{
		printf("%lf ",hx[i]);
		printf(" aaa ");
	}
	printf("\n");
	hipFree(dx);
//	free(hx);
	hipFree(hx);


	return 0;
}


